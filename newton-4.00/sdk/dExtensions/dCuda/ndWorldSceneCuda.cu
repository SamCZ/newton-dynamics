#include "hip/hip_runtime.h"
/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include <ndWorld.h>
#include <ndModel.h>
#include <ndWorldScene.h>
#include <ndBodyDynamic.h>
#include <ndSkeletonList.h>
#include <ndDynamicsUpdate.h>
#include <ndBodyParticleSet.h>
#include <ndDynamicsUpdateSoa.h>
#include <ndJointBilateralConstraint.h>

#include "cuQuat.h"
#include "cuVector.h"
#include "cuMatrix3x3.h"

#include "ndCudaContext.h"
#include "ndWorldSceneCuda.h"
#include "cuSortBodyAabbCells.h"

#define D_CUDA_SCENE_GRID_SIZE		8.0f
#define D_CUDA_SCENE_INV_GRID_SIZE	(1.0f/D_CUDA_SCENE_GRID_SIZE) 

template <typename Predicate>
__global__ void CudaInitBodyArray(Predicate UpdateBodyScene, cuSceneInfo& info)
{
	UpdateBodyScene(info);
}

template <typename Predicate>
__global__ void CudaMergeAabb(Predicate ReducedAabb, cuSceneInfo& info)
{
	ReducedAabb(info);
}

template <typename Predicate>
__global__ void CudaCountAabb(Predicate CountAabb, cuSceneInfo& info)
{
	CountAabb(info);
}

template <typename Predicate>
__global__ void CudaPrefixScanSum0(Predicate PrefixScan, cuSceneInfo& info)
{
	PrefixScan(info);
}

template <typename Predicate>
__global__ void CudaPrefixScanSum1(Predicate PrefixScan, cuSceneInfo& info)
{
	PrefixScan(info);
}

template <typename Predicate>
__global__ void CudaGenerateGridHash(Predicate GenerateHash, cuSceneInfo& info)
{
	if (info.m_frameIsValid)
	{
		GenerateHash(info);
	}
}

template <typename Predicate>
__global__ void CudaEndGridHash(Predicate EndGridHash, cuSceneInfo& info)
{
	if (info.m_frameIsValid)
	{
		EndGridHash(info);
	}
}

template <typename Predicate>
__global__ void CudaGetBodyTransforms(Predicate GetTransform, cuSceneInfo& info, int frameCount)
{
	GetTransform(info, frameCount);
}

template <typename Predicate>
__global__ void CudaInitTransforms(Predicate InitTransforms, cuSceneInfo& info)
{
	InitTransforms(info);
}

__global__ void CudaEndFrame(cuSceneInfo& info, int frameCount)
{
	info.m_frameCount = frameCount;
}

ndWorldSceneCuda::ndWorldSceneCuda(const ndWorldScene& src)
	:ndWorldScene(src)
	,m_context(ndCudaContext::CreateContext())
{
	m_bodyListChanged = 1;
}

ndWorldSceneCuda::~ndWorldSceneCuda()
{
	if (m_context)
	{
		delete m_context;
	}
}

bool ndWorldSceneCuda::IsValid() const
{
	return m_context ? true : false;
}

void ndWorldSceneCuda::Begin()
{
	ndWorldScene::Begin();
	hipDeviceSynchronize();

	hipStream_t stream = m_context->m_solverMemCpyStream;
	const ndInt32 frameCounter = m_context->m_frameCounter;

	// get the scene info from the update	
	cuSceneInfo* const gpuInfo = m_context->m_sceneInfoGpu;
	cuSceneInfo* const cpuInfo = m_context->m_sceneInfoCpu;
	
	hipError_t cudaStatus = hipMemcpyAsync(cpuInfo, gpuInfo, sizeof(cuSceneInfo), hipMemcpyDeviceToHost, stream);
	dAssert(cudaStatus == hipSuccess);
	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}

	CudaEndFrame << < 1, 1, 0, m_context->m_solverComputeStream >> > (*gpuInfo, frameCounter);
	if (frameCounter)
	{
		cuHostBuffer<cuSpatialVector>& cpuBuffer = m_context->m_transformBufferCpu0;
		cuDeviceBuffer<cuSpatialVector>& gpuBuffer = (frameCounter & 1) ? m_context->m_transformBufferGpu1 : m_context->m_transformBufferGpu0;
		gpuBuffer.WriteData(&cpuBuffer[0], cpuBuffer.GetCount() - 1, stream);
	}
}

void ndWorldSceneCuda::End()
{
	m_context->m_frameCounter = m_context->m_frameCounter + 1;
	m_context->SwapBuffers();
	ndWorldScene::End();
}

//void ndWorldSceneCuda::FindCollidingPairs(ndBodyKinematic* const body)
void ndWorldSceneCuda::FindCollidingPairs(ndBodyKinematic* const)
{
	dAssert(0);
}

void ndWorldSceneCuda::FindCollidingPairs()
{
	//ndWorldScene::FindCollidingPairs();
}

//void ndWorldSceneCuda::CalculateContacts(ndInt32 threadIndex, ndContact* const contact)
void ndWorldSceneCuda::CalculateContacts(ndInt32, ndContact* const)
{
	dAssert(0);
}

void ndWorldSceneCuda::CalculateContacts()
{
	//ndWorldScene::CalculateContacts();
}

void ndWorldSceneCuda::LoadBodyData()
{
	auto CopyBodies = ndMakeObject::ndFunction([this](ndInt32 threadIndex, ndInt32 threadCount)
	{
		D_TRACKTIME();
		const ndVector minBox(ndFloat32(1.0e15f));
		const ndVector maxBox(ndFloat32(-1.0e15f));

		ndArray<cuBodyProxy>& data = m_context->m_bodyBufferCpu;
		cuHostBuffer<cuSpatialVector>& transformBufferCpu0 = m_context->m_transformBufferCpu0;
		cuHostBuffer<cuSpatialVector>& transformBufferCpu1 = m_context->m_transformBufferCpu1;

		ndArray<ndBodyKinematic*>& bodyArray = GetActiveBodyArray();
		const ndStartEnd startEnd(bodyArray.GetCount(), threadIndex, threadCount);
		for (ndInt32 i = startEnd.m_start; i < startEnd.m_end; ++i)
		{
			cuSpatialVector transform;
			ndBodyKinematic* const body = bodyArray[i];
			cuBodyProxy& proxi = data[i];

			// Get thansform and velocity
			proxi.m_mass = body->GetMassMatrix();
			proxi.m_rotation = cuQuat(body->GetRotation());
			proxi.m_posit = body->GetGlobalGetCentreOfMass();
			proxi.m_invIntertia = body->GetInvInertia();
			proxi.m_dampCoef = body->GetCachedDamping();
			proxi.m_veloc = body->GetVelocity();
			proxi.m_omega = body->GetOmega();

			// Get scene manager data
			const ndShapeInstance& collision = body->GetCollisionShape();
			const ndShape* const shape = collision.GetShape();

			proxi.m_minAabb = minBox;
			proxi.m_maxAabb = maxBox;
			proxi.m_obbSize = shape->GetObbSize();
			proxi.m_obbOrigin = shape->GetObbOrigin();
			proxi.m_scale = collision.GetScale();
			proxi.m_localPosition = collision.GetLocalMatrix().m_posit;
			proxi.m_localRotation = cuQuat(ndQuaternion(collision.GetLocalMatrix()));
			proxi.m_alignRotation = cuQuat(ndQuaternion(collision.GetAlignmentMatrix()));

			transform.m_angular = cuQuat(body->GetRotation());
			transform.m_linear = body->GetGlobalGetCentreOfMass();
			transformBufferCpu0[i] = transform;
			transformBufferCpu1[i] = transform;
		}
	});

	auto InitTransforms = [] __device__(const cuSceneInfo & info)
	{
		int index = threadIdx.x + blockDim.x * blockIdx.x;
		if (index < info.m_bodyArray.m_size)
		{
			cuBodyProxy* src = info.m_bodyArray.m_array;
			cuSpatialVector* dst0 = info.m_transformBuffer0.m_array;
			cuSpatialVector* dst1 = info.m_transformBuffer1.m_array;

			dst0[index].m_linear = src[index].m_posit;
			dst0[index].m_angular = src[index].m_rotation;
			dst1[index].m_linear = src[index].m_posit;
			dst1[index].m_angular = src[index].m_rotation;
		}
	};

	hipDeviceSynchronize();

	const ndArray<ndBodyKinematic*>& bodyArray = GetActiveBodyArray();

	const ndInt32 cpuBodyCount = bodyArray.GetCount();
	const ndInt32 blocksCount = (cpuBodyCount + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
	const ndInt32 gpuBodyCount = (D_THREADS_PER_BLOCK * ((cpuBodyCount + D_THREADS_PER_BLOCK - 1)) / D_THREADS_PER_BLOCK);

	ndArray<cuBodyProxy>& bodyBufferCpu = m_context->m_bodyBufferCpu;
	bodyBufferCpu.SetCount(cpuBodyCount);

	cuDeviceBuffer<int>& scanGpu = m_context->m_scan;
	cuDeviceBuffer<int>& histogramGpu = m_context->m_histogram;
	cuDeviceBuffer<cuBodyProxy>& bodyBufferGpu = m_context->m_bodyBufferGpu;
	cuDeviceBuffer<cuBoundingBox>& boundingBoxGpu = m_context->m_boundingBoxGpu;
	cuDeviceBuffer<cuBodyAabbCell>& bodyAabbCellGpu0 = m_context->m_bodyAabbCell;
	cuDeviceBuffer<cuBodyAabbCell>& bodyAabbCellGpu1 = m_context->m_bodyAabbCellTmp;
	cuHostBuffer<cuSpatialVector>& transformBufferCpu0 = m_context->m_transformBufferCpu0;
	cuHostBuffer<cuSpatialVector>& transformBufferCpu1 = m_context->m_transformBufferCpu1;
	cuDeviceBuffer<cuSpatialVector>& transformBufferGpu0 = m_context->m_transformBufferGpu0;
	cuDeviceBuffer<cuSpatialVector>& transformBufferGpu1 = m_context->m_transformBufferGpu1;

	scanGpu.SetCount(cpuBodyCount);
	histogramGpu.SetCount(cpuBodyCount);
	bodyBufferGpu.SetCount(cpuBodyCount);
	bodyAabbCellGpu0.SetCount(cpuBodyCount);
	bodyAabbCellGpu1.SetCount(cpuBodyCount);
	transformBufferGpu0.SetCount(cpuBodyCount);
	transformBufferGpu1.SetCount(cpuBodyCount);
	transformBufferCpu0.SetCount(cpuBodyCount);
	transformBufferCpu1.SetCount(cpuBodyCount);
	boundingBoxGpu.SetCount(gpuBodyCount / D_THREADS_PER_BLOCK);

	cuSceneInfo info;
	info.m_scan = cuBuffer<int>(scanGpu);
	info.m_histogram = cuBuffer<int>(histogramGpu);
	info.m_bodyArray = cuBuffer<cuBodyProxy>(bodyBufferGpu);
	info.m_bodyAabbArray = cuBuffer<cuBoundingBox>(boundingBoxGpu);
	info.m_bodyAabbCell = cuBuffer<cuBodyAabbCell>(bodyAabbCellGpu0);
	info.m_bodyAabbCellScrath = cuBuffer<cuBodyAabbCell>(bodyAabbCellGpu1);
	info.m_transformBuffer0 = cuBuffer<cuSpatialVector>(transformBufferGpu0);
	info.m_transformBuffer1 = cuBuffer<cuSpatialVector>(transformBufferGpu1);

	hipError_t cudaStatus;
	ParallelExecute(CopyBodies);

	*m_context->m_sceneInfoCpu = info;
	cudaStatus = hipMemcpy(m_context->m_sceneInfoGpu, &info, sizeof(cuSceneInfo), hipMemcpyHostToDevice);
	dAssert(cudaStatus == hipSuccess);

	bodyBufferGpu.ReadData(&bodyBufferCpu[0], cpuBodyCount);
	CudaInitTransforms << <blocksCount, D_THREADS_PER_BLOCK, 0, 0 >> > (InitTransforms, *m_context->m_sceneInfoCpu);

	hipDeviceSynchronize();

	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}
}

void ndWorldSceneCuda::GetBodyTransforms()
{
	D_TRACKTIME();

	auto GetTransform = [] __device__(const cuSceneInfo& info, int frameCount)
	{
		int index = threadIdx.x + blockDim.x * blockIdx.x;
		if (index < (info.m_bodyArray.m_size - 1))
		{
			cuBodyProxy* src = info.m_bodyArray.m_array;
			cuSpatialVector* dst = (frameCount & 1) ? info.m_transformBuffer0.m_array : info.m_transformBuffer1.m_array;

			dst[index].m_linear = src[index].m_posit;
			dst[index].m_angular = src[index].m_rotation;
		}
	};

	hipStream_t stream = m_context->m_solverComputeStream;
	cuSceneInfo* const infoGpu = m_context->m_sceneInfoGpu;
	
	ndInt32 threads = m_context->m_bodyBufferGpu.GetCount() - 1;
	ndInt32 blocks = (threads + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;
	CudaGetBodyTransforms << <blocks, D_THREADS_PER_BLOCK, 0, stream >> > (GetTransform, *infoGpu, m_context->m_frameCounter);

	//cuHostBuffer<cuSpatialVector>& cpuBuffer = m_context->m_transformBufferCpu0;
	//cuDeviceBuffer<cuSpatialVector>& gpuBuffer = m_context->m_transformBufferGpu0;
	//gpuBuffer.WriteData(&cpuBuffer[0], cpuBuffer.GetCount() - 1, stream);
}

void ndWorldSceneCuda::UpdateTransform()
{
	D_TRACKTIME();

	GetBodyTransforms();
	auto SetTransform = ndMakeObject::ndFunction([this](ndInt32 threadIndex, ndInt32 threadCount)
	{
		D_TRACKTIME();
		const ndArray<ndBodyKinematic*>& bodyArray = GetActiveBodyArray();
		const cuSpatialVector* const data = &m_context->m_transformBufferCpu1[0];
		const ndStartEnd startEnd(bodyArray.GetCount() - 1, threadIndex, threadCount);
		for (ndInt32 i = startEnd.m_start; i < startEnd.m_end; ++i)
		{
			ndBodyKinematic* const body = bodyArray[i];
			const cuSpatialVector& transform = data[i];
			const ndVector position(transform.m_linear.x, transform.m_linear.y, transform.m_linear.z, ndFloat32(1.0f));
			const ndQuaternion rotation(ndVector(transform.m_angular.x, transform.m_angular.y, transform.m_angular.z, transform.m_angular.w));
			body->SetMatrixAndCentreOfMass(rotation, position);

			body->m_transformIsDirty = true;
			UpdateTransformNotify(threadIndex, body);
		}
	});
	ParallelExecute(SetTransform);
}

void ndWorldSceneCuda::UpdateBodyList()
{
	D_TRACKTIME();
	bool bodyListChanged = m_bodyListChanged;
	ndWorldScene::UpdateBodyList();
	if (bodyListChanged)
	{
		LoadBodyData();
	}
	
	cuSceneInfo* const sceneInfo = m_context->m_sceneInfoCpu;
	if (!sceneInfo->m_frameIsValid)
	{
		hipDeviceSynchronize();
		sceneInfo->m_frameIsValid = 1;

		CudaBodyAabbCellResizeBuffers(m_context);

		hipError_t cudaStatus = hipMemcpy(m_context->m_sceneInfoGpu, sceneInfo, sizeof(cuSceneInfo), hipMemcpyHostToDevice);
		dAssert(cudaStatus == hipSuccess);
		if (cudaStatus != hipSuccess)
		{
			dAssert(0);
		}
		hipDeviceSynchronize();
	}
}

void ndWorldSceneCuda::InitBodyArray()
{
//	ndWorldScene::InitBodyArray();

	D_TRACKTIME();
	// this has to be recreated in gpu
	//ndInt32 scans[D_MAX_THREADS_COUNT][2];
	//auto BuildBodyArray = ndMakeObject::ndFunction([this, &scans](ndInt32 threadIndex, ndInt32 threadCount)
	//{
	//	D_TRACKTIME();
	//	const ndArray<ndBodyKinematic*>& view = GetActiveBodyArray();
	//
	//	ndInt32* const scan = &scans[threadIndex][0];
	//	scan[0] = 0;
	//	scan[1] = 0;
	//
	//	const ndFloat32 timestep = m_timestep;
	//	const ndStartEnd startEnd(view.GetCount() - 1, threadIndex, threadCount);
	//	for (ndInt32 i = startEnd.m_start; i < startEnd.m_end; ++i)
	//	{
	//		ndBodyKinematic* const body = view[i];
	//		body->ApplyExternalForces(threadIndex, timestep);
	//
	//		body->PrepareStep(i);
	//		UpdateAabb(threadIndex, body);
	//
	//		const ndInt32 key = body->m_sceneEquilibrium;
	//		scan[key] ++;
	//	}
	//});
	
	//auto CompactMovingBodies = ndMakeObject::ndFunction([this, &scans](ndInt32 threadIndex, ndInt32 threadCount)
	//{
	//	D_TRACKTIME();
	//	const ndArray<ndBodyKinematic*>& activeBodyArray = GetActiveBodyArray();
	//	ndBodyKinematic** const sceneBodyArray = &m_sceneBodyArray[0];
	//
	//	const ndArray<ndBodyKinematic*>& view = m_bodyList.m_view;
	//	ndInt32* const scan = &scans[threadIndex][0];
	//
	//	const ndStartEnd startEnd(view.GetCount(), threadIndex, threadCount);
	//	for (ndInt32 i = startEnd.m_start; i < startEnd.m_end; ++i)
	//	{
	//		ndBodyKinematic* const body = activeBodyArray[i];
	//		const ndInt32 key = body->m_sceneEquilibrium;
	//		const ndInt32 index = scan[key];
	//		sceneBodyArray[index] = body;
	//		scan[key] ++;
	//	}
	//});
	
	//ParallelExecute(BuildBodyArray);
	//ndInt32 sum = 0;
	//ndInt32 threadCount = GetThreadCount();
	//for (ndInt32 j = 0; j < 2; j++)
	//{
	//	for (ndInt32 i = 0; i < threadCount; ++i)
	//	{
	//		const ndInt32 count = scans[i][j];
	//		scans[i][j] = sum;
	//		sum += count;
	//	}
	//}
	//
	//ndInt32 movingBodyCount = scans[0][1] - scans[0][0];
	//m_sceneBodyArray.SetCount(m_bodyList.GetCount());
	//if (movingBodyCount)
	//{
	//	ParallelExecute(CompactMovingBodies);
	//}
	//
	//m_sceneBodyArray.SetCount(movingBodyCount);
	//
	//ndBodyKinematic* const sentinelBody = m_sentinelBody;
	//sentinelBody->PrepareStep(GetActiveBodyArray().GetCount() - 1);
	//
	//sentinelBody->m_isStatic = 1;
	//sentinelBody->m_autoSleep = 1;
	//sentinelBody->m_equilibrium = 1;
	//sentinelBody->m_equilibrium0 = 1;
	//sentinelBody->m_isJointFence0 = 1;
	//sentinelBody->m_isJointFence1 = 1;
	//sentinelBody->m_isConstrained = 0;
	//sentinelBody->m_sceneEquilibrium = 1;
	//sentinelBody->m_weigh = ndFloat32(0.0f);

	auto CalcuateBodyAabb = [] __device__(cuSceneInfo& info)
	{
		__shared__  cuBoundingBox cacheAabb[D_THREADS_PER_BLOCK];

		int threadId = threadIdx.x;
		int index = threadId + blockDim.x * blockIdx.x;
		const int bodyCount = info.m_bodyArray.m_size - 1;
		if (index < bodyCount)
		{
			cuBodyProxy* bodyArray = info.m_bodyArray.m_array;
			cuBodyProxy& body = bodyArray[index];

			// calculate shape global Matrix
			body.m_globalSphapeRotation = body.m_localRotation * body.m_rotation;
			cuMatrix3x3 matrix(body.m_globalSphapeRotation.GetMatrix3x3());
			body.m_globalSphapePosition = matrix.RotateVector(body.m_localPosition) + body.m_posit;

			matrix.m_front = matrix.m_front.Scale(body.m_scale.x);
			matrix.m_up = matrix.m_up.Scale(body.m_scale.y);
			matrix.m_right = matrix.m_right.Scale(body.m_scale.z);
			matrix = body.m_alignRotation.GetMatrix3x3() * matrix;

			const cuVector origin(matrix.RotateVector(body.m_obbOrigin) + body.m_globalSphapePosition);
			const cuVector size(matrix.m_front.Abs().Scale(body.m_obbSize.x) + matrix.m_up.Abs().Scale(body.m_obbSize.y) + matrix.m_right.Abs().Scale(body.m_obbSize.z));

			const cuVector padding(1.0f / 16.0f);
			const cuVector minBox(origin - size - padding);
			const cuVector maxBox(origin + size + padding);

			// save aabb and calculate bonding box for this thread block
			body.m_minAabb = minBox;
			body.m_maxAabb = maxBox;
			cacheAabb[threadId].m_min = minBox;
			cacheAabb[threadId].m_max = maxBox;
		}

		const int lastBlock = bodyCount / D_THREADS_PER_BLOCK;
		if (lastBlock == blockIdx.x)
		{
			__syncthreads();
			const int lastId = bodyCount - D_THREADS_PER_BLOCK * lastBlock;
			const cuBoundingBox box(cacheAabb[0]);
			if (threadId >= lastId)
			{
				cacheAabb[threadId] = box;
			}
		}
		__syncthreads();

		cuBoundingBox* bBox = info.m_bodyAabbArray.m_array;
		for (int i = D_THREADS_PER_BLOCK / 2; i; i = i >> 1)
		{
			if (threadId < i)
			{
				cacheAabb[threadId].m_min = cacheAabb[threadId].m_min.Min(cacheAabb[threadId + i].m_min);
				cacheAabb[threadId].m_max = cacheAabb[threadId].m_max.Max(cacheAabb[threadId + i].m_max);
			}
			__syncthreads();
		}
		
		if (threadId == 0)
		{
			bBox[blockIdx.x].m_min = cacheAabb[0].m_min;
			bBox[blockIdx.x].m_max = cacheAabb[0].m_max;
		}
	};

	auto ReducedAabb = [] __device__(cuSceneInfo& info)
	{
		__shared__  cuBoundingBox cacheAabb[D_THREADS_PER_BLOCK];

		cuBoundingBox* bBoxOut = info.m_bodyAabbArray.m_array;

		int index = threadIdx.x;
		const int boxCount = info.m_bodyAabbArray.m_size;
		if (index < boxCount)
		{
			cacheAabb[index] = bBoxOut[index];
		}
		__syncthreads();

		if (index >= boxCount)
		{
			cacheAabb[index] = cacheAabb[0];
		}
		__syncthreads();

		for (int i = D_THREADS_PER_BLOCK / 2; i; i = i >> 1)
		{
			if (index < i)
			{
				cacheAabb[index].m_min = cacheAabb[index].m_min.Min(cacheAabb[index + i].m_min);
				cacheAabb[index].m_max = cacheAabb[index].m_max.Max(cacheAabb[index + i].m_max);
			}
			__syncthreads();
		}

		if (threadIdx.x == 0)
		{
			cuVector minBox((cacheAabb[0].m_min.Scale(D_CUDA_SCENE_INV_GRID_SIZE).Floor()).Scale(D_CUDA_SCENE_GRID_SIZE));
			cuVector maxBox((cacheAabb[0].m_max.Scale(D_CUDA_SCENE_INV_GRID_SIZE).Floor()).Scale(D_CUDA_SCENE_GRID_SIZE) + cuVector(D_CUDA_SCENE_GRID_SIZE));
			minBox.w = 0.0f;
			maxBox.w = 0.0f;
			const cuVector sizeBox((maxBox - minBox).Scale(D_CUDA_SCENE_INV_GRID_SIZE));
			info.m_hasUpperByteHash.x = (sizeBox.x >= 256);
			info.m_hasUpperByteHash.y = (sizeBox.y >= 256);
			info.m_hasUpperByteHash.z = (sizeBox.z >= 256);

			info.m_worldBox.m_min = minBox;
			info.m_worldBox.m_max = maxBox;
		}
	};

	auto CountAabb = [] __device__(const cuSceneInfo& info)
	{
		int index = threadIdx.x + blockDim.x * blockIdx.x;
		const int bodyCount = info.m_bodyArray.m_size - 1;
		if (index < bodyCount)
		{
			int* scan = info.m_scan.m_array;
			cuBodyProxy* bodyArray = info.m_bodyArray.m_array;

			const cuVector minBox(info.m_worldBox.m_min);
			const cuVector bodyBoxMin(bodyArray[index].m_minAabb);
			const cuVector bodyBoxMax(bodyArray[index].m_maxAabb);

			const int x0 = __float2int_rd((bodyBoxMin.x - minBox.x) * D_CUDA_SCENE_INV_GRID_SIZE);
			const int y0 = __float2int_rd((bodyBoxMin.y - minBox.y) * D_CUDA_SCENE_INV_GRID_SIZE);
			const int z0 = __float2int_rd((bodyBoxMin.z - minBox.z) * D_CUDA_SCENE_INV_GRID_SIZE);
			const int x1 = __float2int_rd((bodyBoxMax.x - minBox.x) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
			const int y1 = __float2int_rd((bodyBoxMax.y - minBox.y) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
			const int z1 = __float2int_rd((bodyBoxMax.z - minBox.z) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
			const int count = (z1 - z0) * (y1 - y0) * (x1 - x0);
			scan[index + 1] = count;
			if (index == 0)
			{
				scan[0] = 0;
			}
		}
	};

	auto PrefixScanSum0 = [] __device__(cuSceneInfo& info)
	{
		__shared__  int cacheBuffer[2 * D_THREADS_PER_BLOCK];

		int threadId = threadIdx.x;
		int threadId1 = threadId + D_THREADS_PER_BLOCK;
		int index = threadId + blockDim.x * blockIdx.x;
		const int bodyCount = info.m_bodyArray.m_size;
		
		int* scan = info.m_scan.m_array;
		cacheBuffer[threadId] = 0;
		cacheBuffer[threadId1] = 0;
		__syncthreads();

		if (index < bodyCount)
		{
			cacheBuffer[threadId1] = scan[index];
		}
		__syncthreads();
		
		for (int i = 1; i < D_THREADS_PER_BLOCK; i = i << 1)
		{
			int sum = cacheBuffer[threadId1] + cacheBuffer[threadId1 - i];
			__syncthreads();
			cacheBuffer[threadId1] = sum;
			__syncthreads();
		}
		if (index < bodyCount)
		{
			scan[index] = cacheBuffer[threadId1];
		}
	};

	auto PrefixScanSum1 = [] __device__(cuSceneInfo& info)
	{
		int threadId = threadIdx.x;
		const int bodyCount = info.m_bodyArray.m_size;
		int* scan = info.m_scan.m_array;

		if ((bodyCount >= D_THREADS_PER_BLOCK) && (bodyCount < D_THREADS_PER_BLOCK * 2))
		{
			int val = scan[D_THREADS_PER_BLOCK - 1];
			int j = D_THREADS_PER_BLOCK + threadId;
			if (j < bodyCount)
			{
				scan[j] = scan[j] + val;
			}
			__syncthreads();
		}
		else
		{
			const int blocks = (bodyCount - 1) / D_THREADS_PER_BLOCK;
			for (int i = 1; i < blocks; i++)
			{
				int sum = scan[i * D_THREADS_PER_BLOCK - 1];
				__syncthreads();
				scan[i * D_THREADS_PER_BLOCK + threadId] += sum;
				__syncthreads();
			}
			int val = scan[blocks * D_THREADS_PER_BLOCK - 1];
			int j = blocks * D_THREADS_PER_BLOCK + threadId;
			if (j < bodyCount)
			{
				scan[j] = scan[j] + val;
			}
			__syncthreads();
		}
		
		if (threadId == 0)
		{
			int newSize = scan[bodyCount - 1];
			info.m_histogram.m_size = newSize;
			info.m_bodyAabbCell.m_size = newSize;
			info.m_bodyAabbCellScrath.m_size = newSize;
		}
	};

	auto GenerateHashGrids = [] __device__(const cuSceneInfo & info)
	{
		const int threadId = threadIdx.x;
		int index = threadId + blockDim.x * blockIdx.x;
		const int bodyCount = info.m_bodyArray.m_size - 1;
		if (index < bodyCount)
		{
			int* scan = info.m_scan.m_array;
			cuBodyProxy* bodyArray = info.m_bodyArray.m_array;
			cuBodyAabbCell* hashArray = info.m_bodyAabbCellScrath.m_array;

			const cuVector minBox(info.m_worldBox.m_min);
			const cuVector bodyBoxMin(bodyArray[index].m_minAabb);
			const cuVector bodyBoxMax(bodyArray[index].m_maxAabb);
			
			const int x0 = __float2int_rd((bodyBoxMin.x - minBox.x) * D_CUDA_SCENE_INV_GRID_SIZE);
			const int y0 = __float2int_rd((bodyBoxMin.y - minBox.y) * D_CUDA_SCENE_INV_GRID_SIZE);
			const int z0 = __float2int_rd((bodyBoxMin.z - minBox.z) * D_CUDA_SCENE_INV_GRID_SIZE);
			const int x1 = __float2int_rd((bodyBoxMax.x - minBox.x) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
			const int y1 = __float2int_rd((bodyBoxMax.y - minBox.y) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;
			const int z1 = __float2int_rd((bodyBoxMax.z - minBox.z) * D_CUDA_SCENE_INV_GRID_SIZE) + 1;

			cuBodyAabbCell hash;
			hash.m_id = index;
			hash.m_key = 0;
			int start = scan[index];
			
			for (int z = z0; z < z1; z++)
			{
				hash.m_z = z;
				for (int y = y0; y < y1; y++)
				{
					hash.m_y = y;
					for (int x = x0; x < x1; x++)
					{
						hash.m_x = x;
						hashArray[start] = hash;
						start++;
					}
				}
			}
		}
	};

	auto EndGridHash = [] __device__(cuSceneInfo& info)
	{
		cuBodyAabbCell* hashArray = info.m_bodyAabbCellScrath.m_array;
		int index = info.m_bodyAabbCellScrath.m_size;

		cuBodyAabbCell hash;
		hash = hashArray[index - 1];
		hash.m_id = unsigned (-1);
		hash.m_z = hash.m_z + 1;

		hashArray[index] = hash;
		info.m_histogram.m_size = index + 1;
		info.m_bodyAabbCell.m_size = index + 1;
		info.m_bodyAabbCellScrath.m_size = index + 1;
		if ((index + 1) >= info.m_bodyAabbCell.m_capacity)
		{
			info.m_frameIsValid = 0;
		}
		if ((2 * index + 2048) >= info.m_histogram.m_capacity)
		{
			info.m_frameIsValid = 0;
		}
	};

	hipStream_t stream = m_context->m_solverComputeStream;
	cuSceneInfo* const infoGpu = m_context->m_sceneInfoGpu;
	
	ndInt32 threads = m_context->m_bodyBufferGpu.GetCount() - 1;
	ndInt32 blocksCount = (threads + D_THREADS_PER_BLOCK - 1) / D_THREADS_PER_BLOCK;

	CudaInitBodyArray << <blocksCount, D_THREADS_PER_BLOCK, 0, stream >> > (CalcuateBodyAabb, *infoGpu);
	CudaMergeAabb << <1, D_THREADS_PER_BLOCK, 0, stream >> > (ReducedAabb, *infoGpu);
	CudaCountAabb << <blocksCount, D_THREADS_PER_BLOCK, 0, stream >> > (CountAabb, *infoGpu);
	CudaPrefixScanSum0 << <blocksCount, D_THREADS_PER_BLOCK, 0, stream >> > (PrefixScanSum0, *infoGpu);
	CudaPrefixScanSum1 << <1, D_THREADS_PER_BLOCK, 0, stream >> > (PrefixScanSum1, *infoGpu);
	CudaGenerateGridHash << <blocksCount, D_THREADS_PER_BLOCK, 0, stream >> > (GenerateHashGrids, *infoGpu);
	CudaEndGridHash << <1, 1, 0, stream >> > (EndGridHash, *infoGpu);
	CudaBodyAabbCellSortBuffer(m_context);
}
