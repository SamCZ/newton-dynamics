/* Copyright (c) <2003-2021> <Julio Jerez, Newton Game Dynamics>
* 
* This software is provided 'as-is', without any express or implied
* warranty. In no event will the authors be held liable for any damages
* arising from the use of this software.
* 
* Permission is granted to anyone to use this software for any purpose,
* including commercial applications, and to alter it and redistribute it
* freely, subject to the following restrictions:
* 
* 1. The origin of this software must not be misrepresented; you must not
* claim that you wrote the original software. If you use this software
* in a product, an acknowledgment in the product documentation would be
* appreciated but is not required.
* 
* 2. Altered source versions must be plainly marked as such, and must not be
* misrepresented as being the original software.
* 
* 3. This notice may not be removed or altered from any source distribution.
*/

#include <ndWorld.h>
#include <ndModel.h>
#include <ndWorldScene.h>
#include <ndBodyDynamic.h>
#include <ndSkeletonList.h>
#include <ndDynamicsUpdate.h>
#include <ndBodyParticleSet.h>
#include <ndDynamicsUpdateSoa.h>
#include <ndJointBilateralConstraint.h>

#include "ndCudaContext.h"
#include "cuSortBodyAabbCells.h"

ndCudaDevice::ndCudaDevice()
{
	hipError_t cudaStatus;
	cudaStatus = hipGetDeviceProperties(&m_prop, 0);
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipSetDevice(0);
	dAssert(cudaStatus == hipSuccess);
	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}
	m_valid = (cudaStatus == hipSuccess);
}

ndCudaDevice::~ndCudaDevice()
{
	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	dAssert(cudaStatus == hipSuccess);

	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}
}

ndCudaContext::ndCudaContext()
	:ndClassAlloc()
	,ndCudaDevice()
	,m_sceneInfoGpu(nullptr)
	,m_sceneInfoCpu(nullptr)
	,m_scan()
	,m_histogram()
	,m_bodyBufferCpu(D_GRANULARITY)
	,m_bodyBufferGpu()
	,m_bodyAabbCell()
	,m_bodyAabbCellTmp()
	,m_boundingBoxGpu()
	,m_transformBufferCpu0()
	,m_transformBufferCpu1()
	,m_transformBufferGpu0()
	,m_transformBufferGpu1()
	,m_solverMemCpyStream(0)
	,m_solverComputeStream(0)
	,m_frameCounter(0)
{
	hipError_t cudaStatus;
	cudaStatus = hipStreamCreate(&m_solverMemCpyStream);
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipStreamCreate(&m_solverComputeStream);
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipMalloc((void**)&m_sceneInfoGpu, sizeof(cuSceneInfo));
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipHostMalloc((void**)&m_sceneInfoCpu, sizeof(cuSceneInfo));
	dAssert(cudaStatus == hipSuccess);

	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}

	*m_sceneInfoCpu = cuSceneInfo();
}

ndCudaContext::~ndCudaContext()
{
	hipError_t cudaStatus;

	cudaStatus = hipHostFree(m_sceneInfoCpu);
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipFree(m_sceneInfoGpu);
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipStreamDestroy(m_solverComputeStream);
	dAssert(cudaStatus == hipSuccess);

	cudaStatus = hipStreamDestroy(m_solverMemCpyStream);
	dAssert(cudaStatus == hipSuccess);

	if (cudaStatus != hipSuccess)
	{
		dAssert(0);
	}
}

ndCudaContext* ndCudaContext::CreateContext()
{
	hipDeviceProp_t prop;
	ndCudaContext* context = nullptr;
	hipError_t cudaStatus = hipGetDeviceProperties(&prop, 0);
	int campbility = prop.major * 10 + prop.minor;
	// go as far back as 5.2 Maxwell GeForce GTX 960 or better.
	if ((cudaStatus == hipSuccess) && (campbility >= 52))
	{
		cudaStatus = hipSetDevice(0);
		context = (cudaStatus == hipSuccess) ? new ndCudaContext() : nullptr;
	}
	return context;
}

void ndCudaContext::SwapBuffers()
{
	m_transformBufferCpu0.Swap(m_transformBufferCpu1);
}